//===------------ debug.cu - NVPTX OpenMP debug utilities -------- CUDA -*-===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
//
// This file contains the implementation of debug utilities to be
// used in the application.
//
//===----------------------------------------------------------------------===//

#include "omptarget-nvptx.h"

////////////////////////////////////////////////////////////////////////////////
// print current state
////////////////////////////////////////////////////////////////////////////////

NOINLINE void PrintTaskDescr(omptarget_nvptx_TaskDescr *taskDescr, char *title,
                             int level) {
  omp_sched_t sched = taskDescr->GetRuntimeSched();
  PRINT(LD_ALL,
        "task descr %s %d: %s, in par %d, dyn %d, rt sched %d, chunk %lld;"
        "  tid %d, tnum %d, nthreads %d\n",
        title, level, (taskDescr->IsParallelConstruct() ? "par" : "task"),
        taskDescr->InParallelRegion(), taskDescr->IsDynamic(), sched,
        taskDescr->RuntimeChunkSize(), taskDescr->ThreadId(),
        taskDescr->ThreadsInTeam(), taskDescr->NThreads());
}

////////////////////////////////////////////////////////////////////////////////
// debug for compiler (should eventually all vanish)
////////////////////////////////////////////////////////////////////////////////

EXTERN void __kmpc_print_str(char *title) { PRINT(LD_ALL, " %s\n", title); }

EXTERN void __kmpc_print_title_int(char *title, int data) {
  PRINT(LD_ALL, "%s val=%d\n", title, data);
}

EXTERN void __kmpc_print_index(char *title, int i) {
  PRINT(LD_ALL, "i = %d\n", i);
}

EXTERN void __kmpc_print_int(int data) { PRINT(LD_ALL, "val=%d\n", data); }

EXTERN void __kmpc_print_double(double data) {
  PRINT(LD_ALL, "val=%lf\n", data);
}

EXTERN void __kmpc_print_address_int64(int64_t data) {
  PRINT(LD_ALL, "val=%016llx\n", data);
}

////////////////////////////////////////////////////////////////////////////////
// substitute for printf in kernel (should vanish)
////////////////////////////////////////////////////////////////////////////////

EXTERN void __kmpc_kernel_print(char *title) { PRINT(LD_ALL, " %s\n", title); }

EXTERN void __kmpc_kernel_print_int8(char *title, int64_t data) {
  PRINT(LD_ALL, " %s val=%lld\n", title, data);
}
